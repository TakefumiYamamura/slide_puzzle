
#include <hip/hip_runtime.h>
#include <stdbool.h>

#undef  SEARCH_ALL_THE_BEST
#undef PACKED /**/
#undef  COLLECT_LOG

#define BLOCK_DIM (32) /* NOTE: broken when more than 32 */
#define N_INIT_DISTRIBUTION (BLOCK_DIM * 64)
#define STACK_BUF_LEN (48 * (BLOCK_DIM/DIR_N))
/* XXX: should be defined dynamically, but cudaMalloc after cudaFree fails */
#define MAX_BUF_RATIO (256)

#define STATE_WIDTH 5
#define STATE_N (STATE_WIDTH * STATE_WIDTH)

typedef unsigned char uchar;
typedef signed char   Direction;
#define dir_reverse(dir) ((Direction)(3 - (dir)))
#define DIR_N 4
#define DIR_FIRST 0
/* this order is not Burns', but Korf's*/
#define DIR_UP 0
#define DIR_LEFT 1
#define DIR_RIGHT 2
#define DIR_DOWN 3
#define POS_X(pos) ((pos) % STATE_WIDTH)
#define POS_Y(pos) ((pos) / STATE_WIDTH)

typedef struct state_tag
{
#ifndef PACKED
    uchar tile[STATE_N];
	uchar inv[STATE_N];
#else
    unsigned long long tile;
#endif
    uchar     empty;
    uchar     depth;
    Direction parent_dir;
	uchar h[4], rh[4];
} d_State;


/* PDB */
#define TABLESIZE 244140625   /* bytes in direct-access database array (25^6) */
static __device__ unsigned char *h0;        /* heuristic tables for pattern databases */
static __device__ unsigned char *h1;

static __device__ __constant__ const int whichpat[25] = {0,0,0,1,1,0,0,0,1,1,2,2,0,1,1,2,2,3,3,3,2,2,3,3,3};
static __device__ __constant__ const int whichrefpat[25] = {0,0,2,2,2,0,0,2,2,2,0,0,0,3,3,1,1,1,3,3,1,1,1,3,3};
#define inv (state->inv)
/* the position of each tile in order, reflected about the main diagonal */
static __device__ __constant__ const int ref[] = {0,5,10,15,20,1,6,11,16,21,2,7,12,17,22,3,8,13,18,23,4,9,14,19,24};
static __device__ __constant__ const int rot90[] = {20,15,10,5,0,21,16,11,6,1,22,17,12,7,2,23,18,13,8,3,24,19,14,9,4};
static __device__ __constant__ const int rot90ref[] = {20,21,22,23,24,15,16,17,18,19,10,11,12,13,14,5,6,7,8,9,0,1,2,3,4};
static __device__ __constant__ const int rot180[] = {24,23,22,21,20,19,18,17,16,15,14,13,12,11,10,9,8,7,6,5,4,3,2,1,0};
static __device__ __constant__ const int rot180ref[] = {24,19,14,9,4,23,18,13,8,3,22,17,12,7,2,21,16,11,6,1,20,15,10,5,0};

static __device__ unsigned int
hash0(d_State *state)
{
	int hashval;                                   /* index into heuristic table */
	hashval = ((((inv[1]*STATE_N+inv[2])*STATE_N+inv[5])*STATE_N+inv[6])*STATE_N+inv[7])*STATE_N+inv[12];
	return (h0[hashval]);                       /* total moves for this pattern */
}

static __device__ unsigned int
hashref0(d_State *state)
{
	int hashval;                                   /* index into heuristic table */
	hashval = (((((ref[inv[5]] * STATE_N + ref[inv[10]]) * STATE_N + ref[inv[1]]) * STATE_N +
					ref[inv[6]]) * STATE_N + ref[inv[11]]) * STATE_N + ref[inv[12]]);
	return (h0[hashval]);                       /* total moves for this pattern */
}

static __device__ unsigned int
hash1(d_State *state)
{
	int hashval;                                   /* index into heuristic table */
	hashval = ((((inv[3]*STATE_N+inv[4])*STATE_N+inv[8])*STATE_N+inv[9])*STATE_N+inv[13])*STATE_N+inv[14];
	return (h1[hashval]);                       /* total moves for this pattern */
}

static __device__ unsigned int
hashref1(d_State *state)
{
	int hashval;                                   /* index into heuristic table */
	hashval = (((((ref[inv[15]] * STATE_N + ref[inv[20]]) * STATE_N + ref[inv[16]]) * STATE_N +
					ref[inv[21]]) * STATE_N + ref[inv[17]]) * STATE_N + ref[inv[22]]);
	return (h1[hashval]);                       /* total moves for this pattern */
}

static __device__ unsigned int
hash2(d_State *state)
{
	int hashval;                                   /* index into heuristic table */
	hashval = ((((rot180[inv[21]] * STATE_N + rot180[inv[20]]) * STATE_N + rot180[inv[16]]) * STATE_N +
				rot180[inv[15]]) * STATE_N + rot180[inv[11]]) * STATE_N + rot180[inv[10]];
	return (h1[hashval]);                       /* total moves for this pattern */
}

static __device__ unsigned int
hashref2(d_State *state)
{
	int hashval;                                   /* index into heuristic table */
	hashval = (((((rot180ref[inv[9]] * STATE_N + rot180ref[inv[4]]) * STATE_N + rot180ref[inv[8]]) * STATE_N +
					rot180ref[inv[3]]) * STATE_N + rot180ref[inv[7]]) * STATE_N + rot180ref[inv[2]]);
	return (h1[hashval]);                       /* total moves for this pattern */
}

static __device__ unsigned int
hash3(d_State *state)
{
	int hashval;                                   /* index into heuristic table */
	hashval = ((((rot90[inv[19]] * STATE_N + rot90[inv[24]]) * STATE_N + rot90[inv[18]]) * STATE_N +
				rot90[inv[23]]) * STATE_N + rot90[inv[17]]) * STATE_N + rot90[inv[22]];
	return (h1[hashval]);                       /* total moves for this pattern */
}

static __device__ unsigned int
hashref3(d_State *state)
{
	int hashval;                                   /* index into heuristic table */
	hashval = (((((rot90ref[inv[23]] * STATE_N + rot90ref[inv[24]]) * STATE_N + rot90ref[inv[18]]) * STATE_N
					+ rot90ref[inv[19]]) * STATE_N + rot90ref[inv[13]]) * STATE_N + rot90ref[inv[14]]);
	return (h1[hashval]);                       /* total moves for this pattern */
}
#undef inv

typedef unsigned int (*HashFunc)(d_State *state);
__device__ HashFunc hash[] = {hash0, hash1, hash2, hash3},
		   rhash[] = {hashref0, hashref1, hashref2, hashref3};


typedef struct search_stat_tag
{
    bool                   solved;
    int                    len;
    unsigned long long int loads;
#ifdef COLLECT_LOG
	unsigned long long int nodes_expanded;
#endif
} search_stat;
typedef struct input_tag
{
    uchar     tiles[STATE_N];
    int       init_depth;
    Direction parent_dir;
} Input;

/* state implementation */

#define state_get_h(s) ((s)->h[0] + (s)->h[1] + (s)->h[2] + (s)->h[3])
#define state_get_rh(s) ((s)->rh[0] + (s)->rh[1] + (s)->rh[2] + (s)->rh[3])
#define state_calc_h(s) (max(state_get_h(s), state_get_rh(s)))
#ifndef PACKED
#define state_tile_get(s, i) ((s)->tile[i])
#define state_tile_set(s, i, v) ((s)->tile[i] = (v))
#define state_inv_set(s, i, v) ((s)->inv[(i)] = (v))

#else
#define STATE_TILE_BITS 4
#define STATE_TILE_MASK ((1ull << STATE_TILE_BITS) - 1)
#define state_tile_ofs(i) (i << 2)
#define state_tile_get(i)                                                      \
    ((state->tile & (STATE_TILE_MASK << state_tile_ofs(i))) >>                 \
     state_tile_ofs(i))
#define state_tile_set(i, val)                                                 \
    do                                                                         \
    {                                                                          \
        state->tile &= ~((STATE_TILE_MASK) << state_tile_ofs(i));              \
        state->tile |= ((unsigned long long) val) << state_tile_ofs(i);        \
    } while (0)
#endif

#define distance(i, j) ((i) > (j) ? (i) - (j) : (j) - (i))
__device__ static void
state_init(d_State *state, Input *input)
{
    state->depth      = input->init_depth;
    state->parent_dir = input->parent_dir;
    for (int i = 0; i < STATE_N; ++i)
    {
        if (input->tiles[i] == 0)
            state->empty = i;
        state_tile_set(state, i, input->tiles[i]);
        state_inv_set(state, input->tiles[i], i);
    }

	for (int i = 0; i < 4; i++)
	{
		state->h[i] = hash[i](state);
		state->rh[i] = rhash[i](state);
	}
}

__device__ static inline bool
state_is_goal(d_State state)
{
    return state_get_h(&state) == 0;
}

__device__ static inline int
state_get_f(d_State state)
{
    return state.depth + state_calc_h(&state);
}

__device__ __shared__ static bool movable_table_shared[STATE_N][DIR_N];

__device__ static inline bool
state_movable(d_State state, Direction dir)
{
    return movable_table_shared[state.empty][dir];
}

__device__ __constant__ const static int pos_diff_table[DIR_N] = {
    -STATE_WIDTH, -1, 1, +STATE_WIDTH};

__device__ static inline bool
state_move(d_State *state, Direction dir, int f_limit)
{
    int new_empty = state->empty + pos_diff_table[dir];
    int opponent  = state_tile_get(state, new_empty);

    state_tile_set(state, state->empty, opponent);
    state_inv_set(state, opponent, state->empty);

	int pat = whichpat[opponent];
	state->h[pat] = hash[pat](state);
	if (state->depth + 1 + state_get_h(state) <= f_limit)
	{
		int rpat = whichrefpat[opponent];
		HashFunc rh;
		if (pat == 0)
			rh = rpat == 0 ? rhash[0] : rhash[2];
		else if (pat == 1)
			rh = rpat == 2 ? rhash[2] : rhash[3];
		else if (pat == 2)
			rh = rpat == 0 ? rhash[0] : rhash[1];
		else
			rh = rpat == 1 ? rhash[1] : rhash[3];
		state->rh[rpat] = rh(state);

		if (state->depth + 1 + state_get_rh(state) <= f_limit)
		{
			state->empty = new_empty;
			state->parent_dir = dir;
			++state->depth;
			return true;
		}
	}

	return false;
}

/* stack implementation */

typedef struct div_stack_tag
{
    unsigned int n;
    d_State      buf[STACK_BUF_LEN];
} d_Stack;

__device__ static inline bool
stack_is_empty(d_Stack *stack)
{
	bool ret = (stack->n == 0);
	__syncthreads();
	return ret;
}

__device__ static inline void
stack_put(d_Stack *stack, d_State *state, bool put)
{
	if (put)
	{
		unsigned int i = atomicInc( &stack->n, UINT_MAX); /* slow? especially in old CC environment */
		stack->buf[i] = *state;
	}
	__syncthreads();
}
__device__ static inline bool
stack_pop(d_Stack *stack, d_State *state)
{
    int tid = threadIdx.x;
    int i   = (int) stack->n - 1 - (int) (tid >> 2);
    if (i >= 0)
        *state = stack->buf[i];
    __syncthreads();
    if (tid == 0)
        stack->n = stack->n >= BLOCK_DIM / DIR_N ?
			stack->n - BLOCK_DIM / DIR_N : 0;
	__syncthreads();
    return i >= 0;
}

//__device__ __shared__ Direction candidate_dir_table[4][3] = {}

/*
 * solver implementation
 */
__device__ static void
idas_internal(d_Stack *stack, int f_limit, search_stat *stat)
{
	d_State state;
    unsigned long long int loop_cnt = 0;
#ifdef COLLECT_LOG
    unsigned long long int nodes_expanded = 0;
#endif
	if (threadIdx.x == 0)
		stat->solved = false;

    for (;;)
    {
        if (stack_is_empty(stack))
		{
			stat->loads = loop_cnt;
#ifdef COLLECT_LOG
			atomicAdd(&stat->nodes_expanded, nodes_expanded);
#endif
			break;
		}

        ++loop_cnt;
        bool found = stack_pop(stack, &state),
			 put = false;

        if (found)
        {
            Direction dir = threadIdx.x & 3;
#ifdef COLLECT_LOG
			nodes_expanded++;
#endif

			/* NOTE: candidate_dir_table may be effective to avoid divergence */
            if (state.parent_dir == dir_reverse(dir))
                continue;

            if (state_movable(state, dir))
            {
                if (state_move(&state, dir, f_limit))
                {
                    if (state_is_goal(state))
					{
#ifndef SEARCH_ALL_THE_BEST
						asm("trap;");
#else
						stat->loads = loop_cnt;
						stat->len = state.depth;
						stat->solved = true;
#endif

#ifdef COLLECT_LOG
						atomicAdd(&stat->nodes_expanded, nodes_expanded);
#endif
					}
                    else
                        put = true;
                }
            }
        }

		stack_put(stack, &state, put);
    }
}

__global__ void
idas_kernel(Input *input, search_stat *stat, int f_limit,
            signed char *h_diff_table, bool *movable_table,
	unsigned char *h0_ptr, unsigned char *h1_ptr, d_Stack *stack_for_all)
{
    //__shared__ d_Stack     stack;
    int tid = threadIdx.x;
	int bid = blockIdx.x;
    d_Stack *stack = &(stack_for_all[bid]);
	if (tid == 0)
{
		h0 = h0_ptr;
		h1 = h1_ptr;
		stat[bid].loads = 0;
}

	d_State state;
	state_init(&state, &input[bid]);
	if (state_get_f(state) > f_limit)
		return;

	if (tid == 0)
	{
		stack->buf[0] = state;
		stack->n      = 1;
	}

    for (int i = tid; i < STATE_N * DIR_N; i += blockDim.x)
        if (i < STATE_N * DIR_N)
            movable_table_shared[i / DIR_N][i % DIR_N] = movable_table[i];

	__syncthreads();
    idas_internal(stack, f_limit, &stat[bid]);
}

/* host library implementation */

#include <errno.h>
#include <limits.h>
#include <stddef.h>
#include <stdio.h>
#include <stdlib.h>

#ifndef UNABLE_LOG
#define elog(...) fprintf(stderr, __VA_ARGS__)
#else
#define elog(...) ;
#endif

void *
palloc(size_t size)
{
    void *ptr = malloc(size);
    if (!ptr)
        elog("malloc failed\n");

    return ptr;
}

void *
repalloc(void *old_ptr, size_t new_size)
{
    void *ptr = realloc(old_ptr, new_size);
    if (!ptr)
        elog("realloc failed\n");

    return ptr;
}

void
pfree(void *ptr)
{
    if (!ptr)
        elog("empty ptr\n");
    free(ptr);
}

#include <assert.h>
#include <stdbool.h>
#include <stdlib.h>
#include <string.h>

typedef unsigned char idx_t;
/*
 *  [0,0] [1,0] [2,0] [3,0]
 *  [0,1] [1,1] [2,1] [3,1]
 *  [0,2] [1,2] [2,2] [3,2]
 *  [0,3] [1,3] [2,3] [3,3]
 */

/*
 * goal state is
 * [0,1,2,3,4,5,6,7,8,9,10,11,12,13,14,15]
 */

typedef struct state_tag_cpu
{
    int       depth; /* XXX: needed? */
    uchar     pos[STATE_WIDTH][STATE_WIDTH];
    idx_t     i, j; /* pos of empty */
    Direction parent_dir;
    int       h_value;
} * State;

#define v(state, i, j) ((state)->pos[i][j])
#define ev(state) (v(state, state->i, state->j))
#define lv(state) (v(state, state->i - 1, state->j))
#define dv(state) (v(state, state->i, state->j + 1))
#define rv(state) (v(state, state->i + 1, state->j))
#define uv(state) (v(state, state->i, state->j - 1))

static uchar from_x[STATE_WIDTH * STATE_WIDTH],
    from_y[STATE_WIDTH * STATE_WIDTH];

static inline void
fill_from_xy(State from)
{
    for (idx_t x = 0; x < STATE_WIDTH; ++x)
        for (idx_t y = 0; y < STATE_WIDTH; ++y)
        {
            from_x[v(from, x, y)] = x;
            from_y[v(from, x, y)] = y;
        }
}

static inline int
heuristic_manhattan_distance(State from)
{
    int h_value = 0;

    fill_from_xy(from);

    for (idx_t i = 1; i < STATE_N; ++i)
    {
        h_value += distance(from_x[i], POS_X(i));
        h_value += distance(from_y[i], POS_Y(i));
    }

    return h_value;
}

bool
state_is_goal(State state)
{
    return state->h_value == 0;
}

static inline State
state_alloc(void)
{
    return (State) palloc(sizeof(struct state_tag_cpu));
}

static inline void
state_free(State state)
{
    pfree(state);
}

State
state_init(uchar v_list[STATE_WIDTH * STATE_WIDTH], int init_depth)
{
    State state = state_alloc();
    int   cnt   = 0;

    state->depth      = init_depth;
    state->parent_dir = (Direction) -1;

    for (idx_t j = 0; j < STATE_WIDTH; ++j)
        for (idx_t i = 0; i < STATE_WIDTH; ++i)
        {
            if (v_list[cnt] == 0)
            {
                state->i = i;
                state->j = j;
            }
            v(state, i, j) = v_list[cnt++];
        }

    state->h_value = heuristic_manhattan_distance(state);

    return state;
}

void
state_fini(State state)
{
    state_free(state);
}

State
state_copy(State src)
{
    State dst = state_alloc();

    memcpy(dst, src, sizeof(*src));

    return dst;
}

static inline bool
state_left_movable(State state)
{
    return state->i != 0;
}
static inline bool
state_down_movable(State state)
{
    return state->j != STATE_WIDTH - 1;
}
static inline bool
state_right_movable(State state)
{
    return state->i != STATE_WIDTH - 1;
}
static inline bool
state_up_movable(State state)
{
    return state->j != 0;
}

bool
state_movable(State state, Direction dir)
{
    return (dir != DIR_LEFT || state_left_movable(state)) &&
           (dir != DIR_DOWN || state_down_movable(state)) &&
           (dir != DIR_RIGHT || state_right_movable(state)) &&
           (dir != DIR_UP || state_up_movable(state));
}

#define h_diff(who, opponent, dir)                                       \
    (h_diff_table[((who) * STATE_N * DIR_N) + ((opponent) << 2) + (dir)])
static int h_diff_table[STATE_N * STATE_N * DIR_N];

void
state_move(State state, Direction dir)
{
    idx_t who;
    assert(state_movable(state, dir));

    switch (dir)
    {
    case DIR_LEFT:
        who = ev(state) = lv(state);
        state->i--;
        break;
    case DIR_DOWN:
        who = ev(state) = dv(state);
        state->j++;
        break;
    case DIR_RIGHT:
        who = ev(state) = rv(state);
        state->i++;
        break;
    case DIR_UP:
        who = ev(state) = uv(state);
        state->j--;
        break;
    default:
        elog("unexpected direction");
        assert(false);
    }

    state->h_value =
        state->h_value + h_diff(who, state->i + state->j * STATE_WIDTH, dir_reverse(dir));
    state->parent_dir = dir;
}

bool
state_pos_equal(State s1, State s2)
{
    for (idx_t i = 0; i < STATE_WIDTH; ++i)
        for (idx_t j = 0; j < STATE_WIDTH; ++j)
            if (v(s1, i, j) != v(s2, i, j))
                return false;

    return true;
}

size_t
state_hash(State state)
{
    /* FIXME: for A* */
    size_t hash_value = 0;
    for (idx_t i = 0; i < STATE_WIDTH; ++i)
        for (idx_t j = 0; j < STATE_WIDTH; ++j)
            hash_value ^= (v(state, i, j) << ((i * 3 + j) << 2));
    return hash_value;
}
int
state_get_hvalue(State state)
{
    return state->h_value;
}

int
state_get_depth(State state)
{
    return state->depth;
}

static void
state_dump(State state)
{
    elog("LOG(state): depth=%d, h=%d, f=%d, ", state->depth, state->h_value,
         state->depth + state->h_value);
    for (int i = 0; i < STATE_N; ++i)
        elog("%d%c", i == state->i + STATE_WIDTH * state->j
                         ? 0
                         : state->pos[i % STATE_WIDTH][i / STATE_WIDTH],
             i == STATE_N - 1 ? '\n' : ',');
}

#include <stddef.h>
#include <stdint.h>
#include <string.h>
#ifndef SIZE_MAX
#define SIZE_MAX ((size_t) -1)
#endif

typedef enum {
    HT_SUCCESS = 0,
    HT_FAILED_FOUND,
    HT_FAILED_NOT_FOUND,
} HTStatus;

/* XXX: hash function for State should be surveyed */
inline static size_t
hashfunc(State key)
{
    return state_hash(key);
}

typedef struct ht_entry_tag *HTEntry;
struct ht_entry_tag
{
    HTEntry next;
    State   key;
    int     value;
};

static HTEntry
ht_entry_init(State key)
{
    HTEntry entry = (HTEntry) palloc(sizeof(*entry));

    entry->key  = state_copy(key);
    entry->next = NULL;

    return entry;
}

static void
ht_entry_fini(HTEntry entry)
{
    pfree(entry);
}

typedef struct ht_tag
{
    size_t   n_bins;
    size_t   n_elems;
    HTEntry *bin;
} * HT;

static bool
ht_rehash_required(HT ht)
{
    return ht->n_bins <= ht->n_elems; /* TODO: local policy is also needed */
}

static size_t
calc_n_bins(size_t required)
{
    /* NOTE: n_bins is used for mask and hence it should be pow of 2, fon now */
    size_t size = 1;
    assert(required > 0);

    while (required > size)
        size <<= 1;

    return size;
}

HT
ht_init(size_t init_size_hint)
{
    size_t n_bins = calc_n_bins(init_size_hint);
    HT     ht     = (HT) palloc(sizeof(*ht));

    ht->n_bins  = n_bins;
    ht->n_elems = 0;

    assert(sizeof(*ht->bin) <= SIZE_MAX / n_bins);
    ht->bin = (HTEntry *) palloc(sizeof(*ht->bin) * n_bins);
    memset(ht->bin, 0, sizeof(*ht->bin) * n_bins);

    return ht;
}

static void
ht_rehash(HT ht)
{
    HTEntry *new_bin;
    size_t   new_size = ht->n_bins << 1;

    assert(ht->n_bins<SIZE_MAX>> 1);

    new_bin = (HTEntry *) palloc(sizeof(*new_bin) * new_size);
    memset(new_bin, 0, sizeof(*new_bin) * new_size);

    for (size_t i = 0; i < ht->n_bins; ++i)
    {
        HTEntry entry = ht->bin[i];

        while (entry)
        {
            HTEntry next = entry->next;

            size_t idx   = hashfunc(entry->key) & (new_size - 1);
            entry->next  = new_bin[idx];
            new_bin[idx] = entry;

            entry = next;
        }
    }

    pfree(ht->bin);
    ht->n_bins = new_size;
    ht->bin    = new_bin;
}

void
ht_fini(HT ht)
{
    for (size_t i = 0; i < ht->n_bins; ++i)
    {
        HTEntry entry = ht->bin[i];
        while (entry)
        {
            HTEntry next = entry->next;
            state_fini(entry->key);
            ht_entry_fini(entry);
            entry = next;
        }
    }

    pfree(ht->bin);
    pfree(ht);
}

HTStatus
ht_insert(HT ht, State key, int **value)
{
    size_t  i;
    HTEntry entry, new_entry;

    if (ht_rehash_required(ht))
        ht_rehash(ht);

    i     = hashfunc(key) & (ht->n_bins - 1);
    entry = ht->bin[i];

    while (entry)
    {
        if (state_pos_equal(key, entry->key))
        {
            *value = &entry->value;
            return HT_FAILED_FOUND;
        }

        entry = entry->next;
    }

    new_entry = ht_entry_init(key);

    new_entry->next = ht->bin[i];
    ht->bin[i]      = new_entry;
    *value          = &new_entry->value;

    assert(ht->n_elems < SIZE_MAX);
    ht->n_elems++;

    return HT_SUCCESS;
}

/*
 * Priority Queue implementation
 */

#include <assert.h>
#include <stdint.h>

typedef struct pq_entry_tag
{
    State state;
    int   f, g;
} PQEntryData;
typedef PQEntryData *PQEntry;

/* tiebreaking is done comparing g value */
static inline bool
pq_entry_higher_priority(PQEntry e1, PQEntry e2)
{
    return e1->f < e2->f || (e1->f == e2->f && e1->g >= e2->g);
}

/*
 * NOTE:
 * This priority queue is implemented doubly reallocated array.
 * It will only extend and will not shrink, for now.
 * It may be improved by using array of layers of iteratively widened array
 */
typedef struct pq_tag
{
    size_t       n_elems;
    size_t       capa;
    PQEntryData *array;
} * PQ;

static inline size_t
calc_init_capa(size_t capa_hint)
{
    size_t capa = 1;
    assert(capa_hint > 0);

    while (capa < capa_hint)
        capa <<= 1;
    return capa - 1;
}

PQ
pq_init(size_t init_capa_hint)
{
    PQ pq = (PQ) palloc(sizeof(*pq));

    pq->n_elems = 0;
    pq->capa    = calc_init_capa(init_capa_hint);

    assert(pq->capa <= SIZE_MAX / sizeof(PQEntryData));
    pq->array = (PQEntryData *) palloc(sizeof(PQEntryData) * pq->capa);

    return pq;
}

void
pq_fini(PQ pq)
{
    for (size_t i = 0; i < pq->n_elems; ++i)
        state_fini(pq->array[i].state);

    pfree(pq->array);
    pfree(pq);
}

static inline bool
pq_is_full(PQ pq)
{
    assert(pq->n_elems <= pq->capa);
    return pq->n_elems == pq->capa;
}

static inline void
pq_extend(PQ pq)
{
    pq->capa = (pq->capa << 1) + 1;
    assert(pq->capa <= SIZE_MAX / sizeof(PQEntryData));

    pq->array =
        (PQEntryData *) repalloc(pq->array, sizeof(PQEntryData) * pq->capa);
}

static inline void
pq_swap_entry(PQ pq, size_t i, size_t j)
{
    PQEntryData tmp = pq->array[i];
    pq->array[i]    = pq->array[j];
    pq->array[j]    = tmp;
}

static inline size_t
pq_up(size_t i)
{
    /* NOTE: By using 1-origin, it may be written more simply, i >> 1 */
    return (i - 1) >> 1;
}

static inline size_t
pq_left(size_t i)
{
    return (i << 1) + 1;
}

static void
heapify_up(PQ pq)
{
    for (size_t i = pq->n_elems; i > 0;)
    {
        size_t ui = pq_up(i);
        assert(i > 0);
        if (!pq_entry_higher_priority(&pq->array[i], &pq->array[ui]))
            break;

        pq_swap_entry(pq, i, ui);
        i = ui;
    }
}

void
pq_put(PQ pq, State state, int f, int g)
{
    if (pq_is_full(pq))
        pq_extend(pq);

    pq->array[pq->n_elems].state = state_copy(state);
    pq->array[pq->n_elems].f     = f; /* this may be abundant */
    pq->array[pq->n_elems].g     = g;
    heapify_up(pq);
    ++pq->n_elems;
}

static void
heapify_down(PQ pq)
{
    size_t sentinel = pq->n_elems;

    for (size_t i = 0;;)
    {
        size_t ri, li = pq_left(i);
        if (li >= sentinel)
            break;

        ri = li + 1;
        if (ri >= sentinel)
        {
            if (pq_entry_higher_priority(&pq->array[li], &pq->array[i]))
                pq_swap_entry(pq, i, li);
            /* Reached the bottom */
            break;
        }

        /* NOTE: If p(ri) == p(li), it may be good to go right
         * since the filling order is left-first */
        if (pq_entry_higher_priority(&pq->array[li], &pq->array[ri]))
        {
            if (!pq_entry_higher_priority(&pq->array[li], &pq->array[i]))
                break;

            pq_swap_entry(pq, i, li);
            i = li;
        }
        else
        {
            if (!pq_entry_higher_priority(&pq->array[ri], &pq->array[i]))
                break;

            pq_swap_entry(pq, i, ri);
            i = ri;
        }
    }
}

State
pq_pop(PQ pq)
{
    State ret_state;

    if (pq->n_elems == 0)
        return NULL;

    ret_state = pq->array[0].state;

    --pq->n_elems;
    pq->array[0] = pq->array[pq->n_elems];
    heapify_down(pq);

    return ret_state;
}

void
pq_dump(PQ pq)
{
    elog("%s: n_elems=%zu, capa=%zu\n", __func__, pq->n_elems, pq->capa);
    for (size_t i = 0, cr_required = 1; i < pq->n_elems; i++)
    {
        if (i == cr_required)
        {
            elog("\n");
            cr_required = (cr_required << 1) + 1;
        }
        elog("%d,", pq->array[i].f);
        elog("%d ", pq->array[i].g);
    }
    elog("\n");
}

#include <stdlib.h>
#include <string.h>

int
rrand(int m)
{
    return (int) ((double) m * (rand() / (RAND_MAX + 1.0)));
}

void
shuffle_input(Input input[], int n_inputs)
{
    Input  tmp;
    size_t n = n_inputs;
    while (n > 1)
    {
        size_t k = rrand(n--);
        memcpy(&tmp, &input[n], sizeof(Input));
        memcpy(&input[n], &input[k], sizeof(Input));
        memcpy(&input[k], &tmp, sizeof(Input));
    }
}

static HT closed;

bool
distribute_astar(State init_state, Input input[], int distr_n, int *cnt_inputs,
                 int *min_fvalue)
{
    int      cnt = 0;
    State    state;
    PQ       q = pq_init(distr_n + 10);
    HTStatus ht_status;
    int *    ht_value;
    bool     solved = false;
    closed          = ht_init(10000);

    ht_status = ht_insert(closed, init_state, &ht_value);
    *ht_value = 0;
    pq_put(q, state_copy(init_state), state_get_hvalue(init_state), 0);
    ++cnt;

    while ((state = pq_pop(q)))
    {
        --cnt;
        if (state_is_goal(state))
        {
            solved = true;
            break;
        }

        ht_status = ht_insert(closed, state, &ht_value);
        if (ht_status == HT_FAILED_FOUND && *ht_value < state_get_depth(state))
        {
            state_fini(state);
            continue;
        }
        else
            *ht_value = state_get_depth(state);

        for (int dir = 0; dir < DIR_N; ++dir)
        {
            if (state->parent_dir != dir_reverse(dir) &&
                state_movable(state, (Direction) dir))
            {
                State next_state = state_copy(state);
                state_move(next_state, (Direction) dir);
                next_state->depth++;

                ht_status = ht_insert(closed, next_state, &ht_value);
                if (ht_status == HT_FAILED_FOUND &&
                    *ht_value <= state_get_depth(next_state))
                    state_fini(next_state);
                else
                {
                    ++cnt;
                    *ht_value = state_get_depth(next_state);
                    pq_put(q, next_state,
                           *ht_value + state_get_hvalue(next_state), *ht_value);
                }
            }
        }

        state_fini(state);

        if (cnt >= distr_n)
            break;
    }

    *cnt_inputs = cnt;
    elog("LOG: init_distr, cnt=%d\n", cnt);
    if (!solved)
    {
        int minf = INT_MAX;
        for (int id = 0; id < cnt; ++id)
        {
            State state = pq_pop(q);
            assert(state);

            for (int i = 0; i < STATE_N; ++i)
                input[id].tiles[i] =
                    state->pos[i % STATE_WIDTH][i / STATE_WIDTH];
            input[id].tiles[state->i + (state->j * STATE_WIDTH)] = 0;

            input[id].init_depth = state_get_depth(state);
            input[id].parent_dir = state->parent_dir;
            if (minf > state_get_depth(state) + state_get_hvalue(state))
                minf = state_get_depth(state) + state_get_hvalue(state);
        }
        assert(pq_pop(q) == NULL);
        // shuffle_input(input, cnt);
        *min_fvalue = minf;
    }

    pq_fini(q);

    return solved;
}

static int
input_devide(Input input[], search_stat stat[], int i, int devide_n, int tail,
             int *buf_len)
{
    int   cnt = 0;
    int * ht_value;
    State state       = state_init(input[i].tiles, input[i].init_depth);
    state->parent_dir = input[i].parent_dir;
    PQ       pq       = pq_init(devide_n);
    HTStatus ht_status;
    pq_put(pq, state, state_get_hvalue(state), 0);
    ++cnt;
    assert(devide_n > 0);

    while ((state = pq_pop(pq)))
    {
        --cnt;
        if (state_is_goal(state))
        {
            /* It may not be optimal goal */
            pq_put(pq, state, state_get_depth(state) + state_get_hvalue(state),
                   state_get_depth(state));
            ++cnt;
            break;
        }

        ht_status = ht_insert(closed, state, &ht_value);
        if (ht_status == HT_FAILED_FOUND && *ht_value < state_get_depth(state))
        {
            state_fini(state);
            continue;
        }
        else
            *ht_value = state_get_depth(state);

        for (int dir = 0; dir < DIR_N; ++dir)
        {
            if (state->parent_dir != dir_reverse(dir) &&
                state_movable(state, (Direction) dir))
            {
                State next_state = state_copy(state);
                state_move(next_state, (Direction) dir);
                next_state->depth++;

                ht_status = ht_insert(closed, next_state, &ht_value);
                if (ht_status == HT_FAILED_FOUND &&
                    *ht_value < state_get_depth(next_state))
                    state_fini(next_state);
                else
                {
                    ++cnt;
                    *ht_value = state_get_depth(next_state);
                    pq_put(pq, next_state,
                           *ht_value + state_get_hvalue(next_state), *ht_value);
                }
            }
        }

        state_fini(state);

        if (cnt >= devide_n)
            break;
    }

    int new_buf_len = *buf_len;
    while (tail + cnt >= new_buf_len)
        new_buf_len <<= 1;
    if (new_buf_len != *buf_len)
    {
        *buf_len = new_buf_len;
        repalloc(input, sizeof(*input) * new_buf_len);
        elog("LOG: host buf resize\n");
    }

    input[i] = input[tail - 1];

    for (int id = 0; id < cnt; ++id)
    {
        int   ofs   = tail - 1 + id;
        State state = pq_pop(pq);
        assert(state);

        for (int j              = 0; j < STATE_N; ++j)
            input[ofs].tiles[j] = state->pos[j % STATE_WIDTH][j / STATE_WIDTH];
        input[ofs].tiles[state->i + (state->j * STATE_WIDTH)] = 0;

        input[ofs].init_depth = state_get_depth(state);
        input[ofs].parent_dir = state->parent_dir;
    }

    pq_fini(pq);

    return cnt - 1;
}

/* main */

#include <errno.h>
#include <stdio.h>
#include <stdlib.h>

#define exit_failure(...)                                                      \
    do                                                                         \
    {                                                                          \
        printf(__VA_ARGS__);                                                   \
        exit(EXIT_FAILURE);                                                    \
    } while (0)

static int
pop_int_from_str(const char *str, char **end_ptr)
{
    long int rv = strtol(str, end_ptr, 0);
    errno       = 0;

    if (errno != 0)
        exit_failure("%s: %s cannot be converted into long\n", __func__, str);
    else if (end_ptr && str == *end_ptr)
        exit_failure("%s: reach end of string", __func__);

    if (rv > INT_MAX || rv < INT_MIN)
        exit_failure("%s: too big number, %ld\n", __func__, rv);

    return (int) rv;
}

#define MAX_LINE_LEN 100
static void
load_state_from_file(const char *fname, uchar *s)
{
    FILE *fp;
    char  str[MAX_LINE_LEN];
    char *str_ptr = str, *end_ptr;

    fp = fopen(fname, "r");
    if (!fp)
        exit_failure("%s: %s cannot be opened\n", __func__, fname);

    if (!fgets(str, MAX_LINE_LEN, fp))
        exit_failure("%s: fgets failed\n", __func__);

    for (int i = 0; i < STATE_N; ++i)
    {
        s[i]    = pop_int_from_str(str_ptr, &end_ptr);
        str_ptr = end_ptr;
    }

    fclose(fp);
}
#undef MAX_LINE_LEN

#define CUDA_CHECK(call)                                                       \
    do                                                                         \
    {                                                                          \
        const hipError_t e = call;                                            \
        if (e != hipSuccess)                                                  \
            exit_failure("Error: %s:%d code:%d, reason: %s\n", __FILE__,       \
                         __LINE__, e, hipGetErrorString(e));                  \
    } while (0)

__host__ static void *
cudaPalloc(size_t size)
{
    void *ptr;
    CUDA_CHECK(hipMalloc(&ptr, size));
    return ptr;
}

__host__ static void
cudaPfree(void *ptr)
{
    CUDA_CHECK(hipFree(ptr));
}

#define h_d_t(op, i, dir)                                                      \
    (h_diff_table[(op) *STATE_N * DIR_N + (i) *DIR_N + (dir)])
__host__ static void
init_mdist(signed char h_diff_table[])
{
    for (int opponent = 0; opponent < STATE_N; ++opponent)
    {
        int goal_x = POS_X(opponent), goal_y = POS_Y(opponent);

        for (int i = 0; i < STATE_N; ++i)
        {
            int from_x = POS_X(i), from_y = POS_Y(i);
            for (uchar dir = 0; dir < DIR_N; ++dir)
            {
                if (dir == DIR_LEFT)
                    h_d_t(opponent, i, dir) = goal_x > from_x ? -1 : 1;
                if (dir == DIR_RIGHT)
                    h_d_t(opponent, i, dir) = goal_x < from_x ? -1 : 1;
                if (dir == DIR_UP)
                    h_d_t(opponent, i, dir) = goal_y > from_y ? -1 : 1;
                if (dir == DIR_DOWN)
                    h_d_t(opponent, i, dir) = goal_y < from_y ? -1 : 1;
            }
        }
    }
}
#undef h_d_t

#define m_t(i, d) (movable_table[(i) *DIR_N + (d)])
__host__ static void
init_movable_table(bool movable_table[])
{
    for (int i = 0; i < STATE_N; ++i)
        for (unsigned int d = 0; d < DIR_N; ++d)
        {
            if (d == DIR_RIGHT)
                m_t(i, d) = (POS_X(i) < STATE_WIDTH - 1);
            else if (d == DIR_LEFT)
                m_t(i, d) = (POS_X(i) > 0);
            else if (d == DIR_DOWN)
                m_t(i, d) = (POS_Y(i) < STATE_WIDTH - 1);
            else if (d == DIR_UP)
                m_t(i, d) = (POS_Y(i) > 0);
        }
}
#undef m_t

static FILE *infile;                              /* pointer to heuristic table file */
static unsigned char h_h0[TABLESIZE];
static unsigned char h_h1[TABLESIZE];
static __host__ void
readfile(unsigned char table[])
{
	int pos[6];                                 /* positions of each pattern tile */
	int index;                                           /* direct access index */

	for (pos[0] = 0; pos[0] < STATE_N; pos[0]++) {
		for (pos[1] = 0; pos[1] < STATE_N; pos[1]++) {
			if (pos[1] == pos[0]) continue;
			for (pos[2] = 0; pos[2] < STATE_N; pos[2]++) {
				if (pos[2] == pos[0] || pos[2] == pos[1]) continue;
				for (pos[3] = 0; pos[3] < STATE_N; pos[3]++) {
					if (pos[3] == pos[0] || pos[3] == pos[1] || pos[3] == pos[2]) continue;
					for (pos[4] = 0; pos[4] < STATE_N; pos[4]++) {
						if (pos[4] == pos[0] || pos[4] == pos[1] || pos[4] == pos[2] || pos[4] == pos[3]) continue;
						for (pos[5] = 0; pos[5] < STATE_N; pos[5]++) {
							if (pos[5] == pos[0] || pos[5] == pos[1] || pos[5] == pos[2] || pos[5] == pos[3] || pos[5] == pos[4])
							continue;
							index = ((((pos[0]*25+pos[1])*25+pos[2])*25+pos[3])*25+pos[4])*25+pos[5];
							table[index] = getc (infile);
						}
					}
				}
			}
		}
	}
}

static __host__ void
pdb_load(void)
{
	infile = fopen("pattern_1_2_5_6_7_12", "rb"); /* read 6-tile pattern database */
	readfile (h_h0);         /* read database and expand into direct-access array */
	fclose(infile);
	printf ("pattern 1 2 5 6 7 12 read in\n");

	infile = fopen("pattern_3_4_8_9_13_14", "rb"); /* read 6-tile pattern database */
	readfile (h_h1);         /* read database and expand into direct-access array */
	fclose(infile);
	printf ("pattern 3 4 8 9 13 14 read in\n");
}

// static char dir_char[] = {'U', 'R', 'L', 'D'};

#define INPUT_SIZE (sizeof(Input) * buf_len)
#define STAT_SIZE (sizeof(search_stat) * buf_len)
#define MOVABLE_TABLE_SIZE (sizeof(bool) * STATE_N * DIR_N)
#define H_DIFF_TABLE_SIZE (STATE_N * STATE_N * DIR_N)
#define INIT_STACK_SIZE (sizeof(d_Stack) * 100000)
int
main(int argc, char *argv[])
{
    int n_roots;

    int buf_len = N_INIT_DISTRIBUTION * MAX_BUF_RATIO;

    Input *input                = (Input *) palloc(INPUT_SIZE),
          *d_input              = (Input *) cudaPalloc(INPUT_SIZE);
    search_stat *stat           = (search_stat *) palloc(STAT_SIZE),
                *d_stat         = (search_stat *) cudaPalloc(STAT_SIZE);
    bool *movable_table         = (bool *) palloc(MOVABLE_TABLE_SIZE),
         *d_movable_table       = (bool *) cudaPalloc(MOVABLE_TABLE_SIZE);
    signed char *h_diff_table   = (signed char *) palloc(H_DIFF_TABLE_SIZE),
                *d_h_diff_table = (signed char *) cudaPalloc(H_DIFF_TABLE_SIZE);
	unsigned char *d_h0 = (unsigned char *) cudaPalloc(TABLESIZE);
	unsigned char *d_h1 = (unsigned char *) cudaPalloc(TABLESIZE);
    d_Stack *stack_for_all = (d_Stack *) cudaPalloc(INIT_STACK_SIZE);

    int min_fvalue = 0;

    if (argc != 2)
        exit_failure("usage: bin/cumain <ifname>\n");

    load_state_from_file(argv[1], input[0].tiles);

	pdb_load();
    CUDA_CHECK(hipMemcpy(d_h0, h_h0, TABLESIZE, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_h1, h_h1, TABLESIZE, hipMemcpyHostToDevice));

    {
        State init_state = state_init(input[0].tiles, 0);
        state_dump(init_state);
        if (distribute_astar(init_state, input, N_INIT_DISTRIBUTION, &n_roots,
                             &min_fvalue))
        {
            elog("solution is found by distributor\n");
            goto solution_found;
        }
        state_fini(init_state);
    }

    init_mdist(h_diff_table);
    init_movable_table(movable_table);

    CUDA_CHECK(hipMemcpy(d_movable_table, movable_table, MOVABLE_TABLE_SIZE,
                          hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_h_diff_table, h_diff_table, H_DIFF_TABLE_SIZE,
                          hipMemcpyHostToDevice));

    CUDA_CHECK(hipMemset(d_input, 0, INPUT_SIZE));

    for (uchar f_limit = min_fvalue;; f_limit += 2)
    {
        CUDA_CHECK(hipMemset(d_stat, 0, STAT_SIZE));
        CUDA_CHECK(
            hipMemcpy(d_input, input, INPUT_SIZE, hipMemcpyHostToDevice));

        elog("f_limit=%d\n", (int) f_limit);
        idas_kernel<<<n_roots, BLOCK_DIM>>>(d_input, d_stat, f_limit,
                                            d_h_diff_table, d_movable_table,
						d_h0, d_h1, stack_for_all);
        CUDA_CHECK(
            hipGetLastError()); /* asm trap is called when find solution */

        CUDA_CHECK(hipMemcpy(stat, d_stat, STAT_SIZE, hipMemcpyDeviceToHost));

        unsigned long long int loads_sum = 0;
        for (int i = 0; i < n_roots; ++i)
            loads_sum += stat[i].loads;

#ifdef COLLECT_LOG
        elog("STAT: loop\n");
        for (int i = 0; i < n_roots; ++i)
            elog("%lld, ", stat[i].loads);
        putchar('\n');
        elog("STAT: nodes_expanded\n");
        for (int i = 0; i < n_roots; ++i)
            elog("%lld, ", stat[i].nodes_expanded);
        putchar('\n');
        elog("STAT: efficiency\n");
        for (int i = 0; i < n_roots; ++i)
		if (stat[i].loads != 0)
            elog("%lld, ", stat[i].nodes_expanded / stat[i].loads);
        putchar('\n');
#endif

        int                    increased = 0;
        unsigned long long int loads_av  = loads_sum / n_roots;

        int stat_cnt[10] = {0, 0, 0, 0, 0, 0, 0, 0, 0};
        for (int i = 0; i < n_roots; ++i)
        {
            if (stat[i].loads < loads_av)
                stat_cnt[0]++;
            else if (stat[i].loads < 2 * loads_av)
                stat_cnt[1]++;
            else if (stat[i].loads < 4 * loads_av)
                stat_cnt[2]++;
            else if (stat[i].loads < 8 * loads_av)
                stat_cnt[3]++;
            else if (stat[i].loads < 16 * loads_av)
                stat_cnt[4]++;
            else if (stat[i].loads < 32 * loads_av)
                stat_cnt[5]++;
            else if (stat[i].loads < 64 * loads_av)
                stat_cnt[6]++;
            else if (stat[i].loads < 128 * loads_av)
                stat_cnt[7]++;
            else
                stat_cnt[8]++;

            int policy = loads_av == 0 ? stat[i].loads
                                       : (stat[i].loads - 1) / loads_av + 1;

            int buf_len_old = buf_len;
            if (policy > 1 && stat[i].loads > 10)
                increased += input_devide(input, stat, i, policy,
                                          n_roots + increased, &buf_len);

            if (buf_len != buf_len_old)
            {
                elog("XXX: fix MAX_BUF_RATIO\n");
                stat = (search_stat *) repalloc(stat, STAT_SIZE);

                cudaPfree(d_input);
                cudaPfree(d_stat);
                d_input = (Input *) cudaPalloc(INPUT_SIZE);
                d_stat  = (search_stat *) cudaPalloc(STAT_SIZE);
            }
        }

        elog("STAT: loads: sum=%lld, av=%lld\n", loads_sum, loads_av);
        elog("STAT: distr: av=%d, 2av=%d, 4av=%d, 8av=%d, 16av=%d, 32av=%d, "
             "64av=%d, 128av=%d, more=%d\n",
             stat_cnt[0], stat_cnt[1], stat_cnt[2], stat_cnt[3], stat_cnt[4],
             stat_cnt[5], stat_cnt[6], stat_cnt[7], stat_cnt[8]);

        n_roots += increased;
        elog("STAT: n_roots=%d(+%d)\n", n_roots, increased);

#ifdef SEARCH_ALL_THE_BEST
        for (int i = 0; i < n_roots; ++i)
            if (stat[i].solved)
            {
                elog("find all the optimal solution(s), at depth=%d\n", stat[i].len);
                goto solution_found;
            }
#endif
    }

solution_found:
    cudaPfree(d_input);
    cudaPfree(d_stat);
    cudaPfree(d_movable_table);
    cudaPfree(d_h_diff_table);
    cudaPfree(d_h0);
    cudaPfree(d_h1);

    CUDA_CHECK(hipDeviceReset());

    pfree(input);
    pfree(stat);
    pfree(movable_table);
    pfree(h_diff_table);

    return 0;
}
